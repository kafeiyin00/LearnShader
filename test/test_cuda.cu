#include "hip/hip_runtime.h"
#include <iostream>
#include "C:/Program Files/NVIDIA GPU Computing Toolkit/CUDA/v10.0/include/hip/hip_runtime.h"
#include "hip/hip_runtime.h"


__global__ void func(float* d_out, float* d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f*f;
}


void getCudaState(){
	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++)
	{

		hipGetDeviceProperties(&prop, i);
		std::cout << "�Կ����ƣ�" << prop.name << std::endl;
		std::cout << "�Դ��С��" << prop.totalGlobalMem / 1024 / 1024 << " MB" << std::endl;
		std::cout << "һ��block�Ĺ����ڴ��С��" << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
		std::cout << "block����߳�����" << prop.maxThreadsPerBlock << std::endl;
	}
}
int main()
{
	getCudaState();


	const int arraySize = 200;
    const int byteSize = arraySize*sizeof(float);

    //cpu
    float h_in[arraySize];
    for (int i = 0; i < arraySize; ++i) {
        h_in[i]=i;
    }
    float h_out[arraySize];

    //GPU
    float * d_in;
    float * d_out;

    //allocate GPU memory
    hipMalloc((void**)&d_in,byteSize);
    hipMalloc((void**)&d_out,byteSize);

    //from cpu to gpu
    hipMemcpy(d_in,h_in,byteSize,hipMemcpyHostToDevice);

    func<<<1,arraySize>>>(d_out,d_in);

    //from gpu to cpu
    hipMemcpy(h_out,d_out,byteSize,hipMemcpyDeviceToHost);


    for (int i = 0; i < arraySize; ++i) {
        std::cout<<h_out[i]<<"\n";
    }

    hipFree(d_in);
    hipFree(d_out);
	return 0;
}